/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/cuda_in_memory_key_value_store.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/embedding/embedding_manager.h"

namespace oneflow {

namespace {}  // namespace

template<typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() = default;
  ~EmbeddingLookupKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    LOG(ERROR) << "EmbeddingLookupKernel";
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());

    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    IDX* host_num_keys;
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys, 1 * sizeof(IDX)));
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    // store->Lookup(stream, *host_num_keys, num_unique_ids->dptr<K>(), context->dptr<K>(),
    //          embeddings->dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(dtype)                   \
  REGISTER_USER_KERNEL("embedding_lookup")                             \
      .SetCreateFn<EmbeddingLookupKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("unique_ids", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(int32_t)
REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(int64_t)

template<typename T>
class EmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  EmbeddingUpdateKernel() = default;
  ~EmbeddingUpdateKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    LOG(ERROR) << "EmbeddingUpdateKernel";
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(dtype)                   \
  REGISTER_USER_KERNEL("sgd_embedding_update")                         \
      .SetCreateFn<EmbeddingUpdateKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("unique_ids", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(int32_t)
REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(int64_t)

}  // namespace oneflow
