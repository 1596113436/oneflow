/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/embedding/cuda_in_memory_key_value_store.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/include/primitive/memcpy.h"
#include "oneflow/core/embedding/embedding_manager.h"

namespace oneflow {

namespace {}  // namespace

template<typename K, typename IDX>
class EmbeddingPrefetchKernel final : public user_op::OpKernel {
 public:
  EmbeddingPrefetchKernel() = default;
  ~EmbeddingPrefetchKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    LOG(ERROR) << "EmbeddingPrefetchKernel";
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    IDX* host_num_keys;
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys, 1 * sizeof(IDX)));
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    uint32_t num_keys = *host_num_keys;
    store->Prefetch(ctx->stream(), num_keys, unique_ids->dptr(),
                    reinterpret_cast<uint64_t*>(context->mut_dptr()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(k_dtype, idx_dtype) \
  REGISTER_USER_KERNEL("embedding_prefetch")                        \
      .SetCreateFn<EmbeddingPrefetchKernel<k_dtype, idx_dtype>>()   \
      .SetIsMatchedHob(                                             \
          (user_op::HobDeviceType() == DeviceType::kCUDA)           \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value));

REGISTER_CUDA_EMBEDDING_PREFETCH_KERNEL(int64_t, int32_t)

template<typename T, typename K, typename IDX>
class EmbeddingLookupKernel final : public user_op::OpKernel {
 public:
  EmbeddingLookupKernel() = default;
  ~EmbeddingLookupKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    LOG(ERROR) << "EmbeddingLookupKernel";
    embedding::KeyValueStore* store = Global<EmbeddingMgr>::Get()->GetKeyValueStore(
        "MyEmbeddingTest", ctx->parallel_ctx().parallel_id());

    hipStream_t hip_stream = ctx->stream()->As<ep::CudaStream>()->hip_stream();
    const user_op::Tensor* num_unique_ids = ctx->Tensor4ArgNameAndIndex("num_unique_ids", 0);
    const user_op::Tensor* unique_ids = ctx->Tensor4ArgNameAndIndex("unique_ids", 0);
    const user_op::Tensor* context = ctx->Tensor4ArgNameAndIndex("context", 0);
    user_op::Tensor* embeddings = ctx->Tensor4ArgNameAndIndex("embeddings", 0);
    IDX* host_num_keys;
    OF_CUDA_CHECK(hipHostMalloc(&host_num_keys, 1 * sizeof(IDX)));
    std::unique_ptr<ep::primitive::Memcpy> copyd2h_primitive =
        ep::primitive::NewPrimitive<ep::primitive::MemcpyFactory>(DeviceType::kCUDA,
                                                                  ep::primitive::MemcpyKind::kDtoH);
    CHECK(copyd2h_primitive);
    copyd2h_primitive->Launch(ctx->stream(), host_num_keys, num_unique_ids->dptr(), sizeof(IDX));
    CHECK_JUST(ctx->stream()->Sync());
    store->Lookup(ctx->stream(), *host_num_keys, unique_ids->dptr(),
                  reinterpret_cast<const uint64_t*>(context->dptr()), embeddings->mut_dptr());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(t_dtype, k_dtype, idx_dtype)                \
  REGISTER_USER_KERNEL("embedding_lookup")                                                \
      .SetCreateFn<EmbeddingLookupKernel<t_dtype, k_dtype, idx_dtype>>()                  \
      .SetIsMatchedHob(                                                                   \
          (user_op::HobDeviceType() == DeviceType::kCUDA)                                 \
          && (user_op::HobDataType("num_unique_ids", 0) == GetDataType<idx_dtype>::value) \
          && (user_op::HobDataType("unique_ids", 0) == GetDataType<k_dtype>::value)       \
          && (user_op::HobDataType("embeddings", 0) == GetDataType<t_dtype>::value));

REGISTER_CUDA_EMBEDDING_LOOKUP_KERNEL(float, int64_t, int32_t)

template<typename T>
class EmbeddingUpdateKernel final : public user_op::OpKernel {
 public:
  EmbeddingUpdateKernel() = default;
  ~EmbeddingUpdateKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    LOG(ERROR) << "EmbeddingUpdateKernel";
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(dtype)                   \
  REGISTER_USER_KERNEL("sgd_embedding_update")                         \
      .SetCreateFn<EmbeddingUpdateKernel<dtype>>()                     \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("unique_ids", 0) == GetDataType<dtype>::value));

REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(int32_t)
REGISTER_CUDA_EMBEDDING_UPDATE_KERNEL(int64_t)

}  // namespace oneflow
