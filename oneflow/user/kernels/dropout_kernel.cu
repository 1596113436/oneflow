#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <cstdint>
#include <memory>
#include "oneflow/user/kernels/dropout_kernel.h"
// #include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/op_kernel_state_wrapper.h"
#include "oneflow/core/kernel/random_generator.h"
// #include "oneflow/user/kernels/random_mask_generator.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/device_type.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/core/kernel/cuda_graph_support.h"

namespace oneflow {

namespace {

using H2PackType = typename std::aligned_storage<4 * sizeof(half), 4 * sizeof(half)>::type;
union H2Pack{
  H2PackType storage; 
  half2 h2[2]; 
};

template<typename T, int pack_size>
__global__ void MaskAndScaleGpu(uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const T* x, int8_t* mask,
                                T* y) {
    uint64_t cur_seed = seed[0]; 
    int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
    hiprandStatePhilox4_32_10_t state; 
    // auto seeds = at::cuda::philox::unpack(philox_args);
    hiprand_init(0, thread_id, 0, &state); 
    using LoadT = typename std::aligned_storage<sizeof(T)*pack_size, sizeof(T)*pack_size>::type; 
    using MaskT = typename std::aligned_storage<sizeof(int8_t)*pack_size, sizeof(int8_t)*pack_size>::type; 

    float4 rand_uniform; 
    for(int64_t linear_idx=thread_id*pack_size; linear_idx < n; linear_idx += gridDim.x * blockDim.x * pack_size) {
      rand_uniform = hiprand_uniform4(&state);
      rand_uniform.x = rand_uniform.x >= rate; 
      rand_uniform.y = rand_uniform.y >= rate; 
      rand_uniform.z = rand_uniform.z >= rate; 
      rand_uniform.w = rand_uniform.w >= rate; 
      const LoadT* x_load = reinterpret_cast<const LoadT*>(&x[linear_idx]);
      cuda::elementwise::Pack<T, pack_size> x_vec;
      x_vec.storage = *x_load; 

      int8_t mask_vec[pack_size];
      T y_vec[pack_size]; 
      #pragma unroll
      for (int i = 0; i < pack_size; i++) {
        mask_vec[i] = (&rand_uniform.x)[i] >= rate;
        y_vec[i] = x_vec.elem[i]*mask_vec[i]*scale;
      }

      *(reinterpret_cast<LoadT*>(y+linear_idx)) = *reinterpret_cast<LoadT*>(y_vec);
      *(reinterpret_cast<MaskT*>(mask+linear_idx)) = *reinterpret_cast<MaskT*>(mask_vec);
    }
    __syncthreads();

    if(thread_id == 0) {
      int32_t new_counter = cuda::atomic::Add(counter, 1) + 1; 
      if(new_counter == gridDim.x) {
        *seed += n;
        *counter = 0;
      }
    }
}


template<typename T, int pack_size>
__global__ void MaskAndScaleAddGpu(uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const T* x, int8_t* mask,
                                   const T* addend, T* y) {
  uint64_t cur_seed = seed[0]; 
  int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
  hiprandStatePhilox4_32_10_t state; 
  // auto seeds = at::cuda::philox::unpack(philox_args);
  hiprand_init(0, thread_id, 0, &state); 
  using LoadT = typename std::aligned_storage<sizeof(T)*pack_size, sizeof(T)*pack_size>::type; 
  using MaskT = typename std::aligned_storage<sizeof(int8_t)*pack_size, sizeof(int8_t)*pack_size>::type; 

  float4 rand_uniform; 
  for(int64_t linear_idx=thread_id*pack_size; linear_idx < n; linear_idx += gridDim.x * blockDim.x * pack_size) {
    rand_uniform = hiprand_uniform4(&state);
    rand_uniform.x = rand_uniform.x >= rate; 
    rand_uniform.y = rand_uniform.y >= rate; 
    rand_uniform.z = rand_uniform.z >= rate; 
    rand_uniform.w = rand_uniform.w >= rate; 
    const LoadT* x_load = reinterpret_cast<const LoadT*>(&x[linear_idx]);
    cuda::elementwise::Pack<T, pack_size> x_vec;
    x_vec.storage = *x_load; 

    const LoadT* addend_load = reinterpret_cast<const LoadT*>(&addend[linear_idx]);
    cuda::elementwise::Pack<T, pack_size> addend_vec;
    addend_vec.storage = *addend_load; 

    int8_t mask_vec[pack_size];
    T y_vec[pack_size]; 
    #pragma unroll
    for (int i = 0; i < pack_size; i++) {
      mask_vec[i] = (&rand_uniform.x)[i] >= rate;
      y_vec[i] = x_vec.elem[i]*mask_vec[i]*scale + addend_vec.elem[i];
    }

    *(reinterpret_cast<LoadT*>(y+linear_idx)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask+linear_idx)) = *reinterpret_cast<MaskT*>(mask_vec);
  }
  __syncthreads();

  if(thread_id == 0) {
    int32_t new_counter = cuda::atomic::Add(counter, 1) + 1; 
    if(new_counter == gridDim.x) {
      *seed += n;
      *counter = 0;
    }
  }
}

template<>
__global__ void MaskAndScaleGpu<half, 4>(uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const half* x, int8_t* mask,
                                half* y) {
    int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
    hiprandStatePhilox4_32_10_t state; 
    // auto seeds = at::cuda::philox::unpack(philox_args);
    hiprand_init(0, thread_id, 0, &state); 
    using LoadT = typename std::aligned_storage<sizeof(half)*4, sizeof(half)*4>::type; 
    using MaskT = typename std::aligned_storage<sizeof(int8_t)*4, sizeof(int8_t)*4>::type; 

    float4 rand_uniform; 
    half2 h2_scale = __float2half2_rn(scale);
    for(int64_t linear_idx=thread_id*4; linear_idx < n; linear_idx += gridDim.x * blockDim.x * 4) {
      rand_uniform = hiprand_uniform4(&state);

      const LoadT* x_load = reinterpret_cast<const LoadT*>(&x[linear_idx]);
      H2Pack x_vec{};
      x_vec.storage = *x_load; 

      int8_t mask_vec[4];
      half2 y_vec[2]; 
      half2 one_or_zero_h2[2];

      mask_vec[0] = (&rand_uniform.x)[0] >= rate;
      one_or_zero_h2[0].x = mask_vec[0]; 
      mask_vec[1] = (&rand_uniform.y)[1] >= rate;
      one_or_zero_h2[0].y = mask_vec[1]; 
      y_vec[0] = __hmul2(__hmul2(x_vec.h2[0], one_or_zero_h2[0]), h2_scale); 

      mask_vec[2] = (&rand_uniform.z)[2] >= rate;
      one_or_zero_h2[1].x = mask_vec[2]; 
      mask_vec[3] = (&rand_uniform.w)[3] >= rate;
      one_or_zero_h2[1].y = mask_vec[3]; 
      y_vec[1] = __hmul2(__hmul2(x_vec.h2[1], one_or_zero_h2[1]), h2_scale); 
      
      *(reinterpret_cast<LoadT*>(y+linear_idx)) = *reinterpret_cast<LoadT*>(y_vec);
      *(reinterpret_cast<MaskT*>(mask+linear_idx)) = *reinterpret_cast<MaskT*>(mask_vec);
    }
    __syncthreads();

    if(thread_id == 0) {
      int32_t new_counter = cuda::atomic::Add(counter, 1) + 1; 
      if(new_counter == gridDim.x) {
        *seed += n;
        *counter = 0;
      }
    }
}

template<>
__global__ void MaskAndScaleAddGpu<half, 4>(uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const half* x, int8_t* mask, const half* addend, half* y) {
    int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
    hiprandStatePhilox4_32_10_t state; 
    // auto seeds = at::cuda::philox::unpack(philox_args);
    hiprand_init(0, thread_id, 0, &state); 
    using LoadT = typename std::aligned_storage<sizeof(half)*4, sizeof(half)*4>::type; 
    using MaskT = typename std::aligned_storage<sizeof(int8_t)*4, sizeof(int8_t)*4>::type; 

    float4 rand_uniform; 
    half2 h2_scale = __float2half2_rn(scale);
    for(int64_t linear_idx=thread_id*4; linear_idx < n; linear_idx += gridDim.x * blockDim.x * 4) {
      rand_uniform = hiprand_uniform4(&state);

      const LoadT* x_load = reinterpret_cast<const LoadT*>(&x[linear_idx]);
      H2Pack x_vec{};
      x_vec.storage = *x_load; 

      const LoadT* addend_load = reinterpret_cast<const LoadT*>(&addend[linear_idx]);
      H2Pack addend_vec{};
      addend_vec.storage = *addend_load; 

      int8_t mask_vec[4];
      half2 y_vec[2]; 
      half2 one_or_zero_h2[2];

      mask_vec[0] = (&rand_uniform.x)[0] >= rate;
      one_or_zero_h2[0].x = mask_vec[0]; 
      mask_vec[1] = (&rand_uniform.y)[1] >= rate;
      one_or_zero_h2[0].y = mask_vec[1]; 
      y_vec[0] = __hadd2(__hmul2(__hmul2(x_vec.h2[0], one_or_zero_h2[0]), h2_scale), addend_vec.h2[0]); 

      mask_vec[2] = (&rand_uniform.z)[2] >= rate;
      one_or_zero_h2[1].x = mask_vec[2]; 
      mask_vec[3] = (&rand_uniform.w)[3] >= rate;
      one_or_zero_h2[1].y = mask_vec[3]; 
      y_vec[1] = __hadd2(__hmul2(__hmul2(x_vec.h2[1], one_or_zero_h2[1]), h2_scale), addend_vec.h2[0]); 
      
      *(reinterpret_cast<LoadT*>(y+linear_idx)) = *reinterpret_cast<LoadT*>(y_vec);
      *(reinterpret_cast<MaskT*>(mask+linear_idx)) = *reinterpret_cast<MaskT*>(mask_vec);
    }
    __syncthreads();

    if(thread_id == 0) {
      int32_t new_counter = cuda::atomic::Add(counter, 1) + 1; 
      if(new_counter == gridDim.x) {
        *seed += n;
        *counter = 0;
      }
    }
}

template<int pack_size>
void ComputeGridSize(const int32_t block_size, unsigned int* grid_size){
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop,0);
  unsigned int blocks_per_sm = prop.maxThreadsPerMultiProcessor/block_size;
  *grid_size = std::min((unsigned int)prop.multiProcessorCount * blocks_per_sm, *grid_size);
}

template<typename T>
void MaskAndScale(DeviceCtx* ctx, uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const T* x, int8_t* mask,
                  T* y) {
  int32_t UNROLL = 4; 
  int32_t block_size = 256; 
  unsigned int grid_size = ((n + block_size -1) / block_size);
  ComputeGridSize<4>(block_size, &grid_size); 
  int64_t counter_offset = ((n - 1)/(block_size*grid_size*UNROLL)+1)*UNROLL;
//   std::lock_guard<std::mutex> lock(generator_->mutex_);
  // one::PhiloxCUDAState rng_engine_inputs = generator_->philox_cuda_state(counter_offset);
  printf("Grid size is: %u \n", grid_size); 
  printf("Block size is: %u \n", block_size); 
  MaskAndScaleGpu<T, 4><<<grid_size, block_size, 0, ctx->hip_stream()>>>(seed, counter, n, rate, scale, x, mask, y);
}


template<typename T>
void MaskAndScaleAdd(DeviceCtx* ctx, uint64_t* seed, int32_t* counter, const int64_t n, float rate, float scale, const T* x, int8_t* mask,
                     const T* addend, T* y) {
  int32_t UNROLL = 4; 
  int32_t block_size = 256; 
  unsigned int grid_size = ((n + block_size -1) / block_size);
  ComputeGridSize<4>(block_size, &grid_size); 
  int64_t counter_offset = ((n - 1)/(block_size*grid_size*UNROLL)+1)*UNROLL;
  // std::lock_guard<std::mutex> lock(generator_->mutex_);
  // one::PhiloxCUDAState rng_engine_inputs = generator_->philox_cuda_state(counter_offset);
  printf("Grid size is: %u \n", grid_size); 
  printf("Block size is: %u \n", block_size); 
  MaskAndScaleAddGpu<T, 4><<<grid_size, block_size, 0, ctx->hip_stream()>>>(seed, counter, n, rate, scale, x, mask, addend, y);
}

template<typename T>
struct MaskAndScaleFunctor {
  OF_DEVICE_FUNC explicit MaskAndScaleFunctor(float scale) : scale(scale) {}
  OF_DEVICE_FUNC T operator()(T x, int8_t mask) const {
    return x * static_cast<T>(mask) * static_cast<T>(scale);
  }
  float scale;
};

template<typename T>
class DropoutKernelGPU final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  DropoutKernelGPU() = default;
  ~DropoutKernelGPU() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* fused_dropout_kernel_state = dynamic_cast<FusedDropoutKernelState*>(state);
    // CHECK_NOTNULL(fused_dropout_kernel_state);
    const auto& generator = fused_dropout_kernel_state->generator();
    // CHECK_NOTNULL(generator);
    std::shared_ptr<one::CUDAGeneratorImpl> cuda_gen = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>());

    const float rate = ctx->Attr<float>("rate");
    float scale = 1.0;
    if (rate != 1.0) { scale = 1.0 / (1.0 - rate); }
    // const auto& generator = CHECK_JUST(one::MakeGenerator(DeviceType::kGPU));
    // generator->set_current_seed(ctx->Attr<int64_t>("seed"));
    // std::shared_ptr<RandomMaskGenerator<DeviceType::kGPU>> random_mask_like_gen = std::make_shared<RandomMaskGenerator<DeviceType::kGPU>>(generator);

    uint64_t* seed = cuda_gen->dev_seed(); 
    int32_t* counter = cuda_gen->dev_counter(); 

    if (ctx->has_input("_add_to_output", 0)) {
      printf("Do nothing skip! \n"); 
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      hipMalloc(&seed, sizeof(uint64_t)); 
      hipMalloc(&counter, sizeof(int32_t)); 
      MaskAndScaleAdd<T>(ctx->device_ctx(), seed, counter, in->shape().elem_cnt(), rate, scale, in->dptr<T>(),
                         mask->mut_dptr<int8_t>(), addend->dptr<T>(), out->mut_dptr<T>());
      hipFree(seed); 
      hipFree(counter); 
    } else {
      hipMalloc(&seed, sizeof(uint64_t)); 
      hipMalloc(&counter, sizeof(int32_t)); 

      MaskAndScale<T>(ctx->device_ctx(), seed, counter, in->shape().elem_cnt(), rate, scale, in->dptr<T>(),
                      mask->mut_dptr<int8_t>(), out->mut_dptr<T>());
      hipFree(seed); 
      hipFree(counter); 
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_KERNEL_GPU(dtype)                                                \
  REGISTER_USER_KERNEL("dropout").SetCreateFn<DropoutKernelGPU<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceTag() == "gpu")                                                  \
      & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value)                  \
      & (user_op::HobDataType("mask", 0) == GetDataType<int8_t>::value));

REGISTER_DROPOUT_KERNEL_GPU(half)
REGISTER_DROPOUT_KERNEL_GPU(float)
REGISTER_DROPOUT_KERNEL_GPU(double)

template<typename T>
class DropoutGradKernelGPU final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  DropoutGradKernelGPU() = default;
  ~DropoutGradKernelGPU() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float scale = ctx->Attr<float>("scale");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    OF_CUDA_CHECK((cuda::elementwise::Binary(MaskAndScaleFunctor<T>(scale), elem_cnt,
                                             dx->mut_dptr<T>(), dy->dptr<T>(), mask->dptr<int8_t>(),
                                             ctx->device_ctx()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_GRAD_KERNEL_GPU(dtype)                                                 \
  REGISTER_USER_KERNEL("dropout_grad")                                                          \
      .SetCreateFn<DropoutGradKernelGPU<dtype>>()                                               \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                       \
                       & (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value))          \
      .SetInplaceProposalFn([](const user_op::InferContext&,                                    \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "dy", 0, true));                        \
        return Maybe<void>::Ok();                                                               \
      });

// REGISTER_DROPOUT_GRAD_KERNEL_GPU(half)
REGISTER_DROPOUT_GRAD_KERNEL_GPU(float)
REGISTER_DROPOUT_GRAD_KERNEL_GPU(double)

}  // namespace

}  // namespace oneflow
