#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/device_type.pb.h"
#include "oneflow/user/kernels/op_kernel_state_wrapper.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/device_type.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/cuda/atomic.cuh"
#include "oneflow/user/kernels/dropout_kernel.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/ep/cuda/hip_stream.h"

namespace oneflow {

namespace {

/*
hiprand_uniform4 interval is (0, 1.0]
*/

constexpr int32_t kVecSize = 4;
constexpr int32_t kBlockSize = 256;
constexpr int32_t PackDoubleSize = 2;
constexpr int32_t PackFloatSize = 4;
constexpr int32_t PackHalfSize = 4;
constexpr int32_t PackHalf2Size = 2;

using H2PackType = typename std::aligned_storage<4 * sizeof(half), 4 * sizeof(half)>::type;
union H2Pack {
  H2PackType storage;
  half2 h2[2];
};

union RandPack4 {
  float4 storage;
  float elem[4];
};

#define RETURN_VOID_IF_HALF typename std::enable_if_t<std::is_same<T, half>::value, void>
#define RETURN_VOID_IF_FLOAT typename std::enable_if_t<std::is_same<T, float>::value, void>
#define RETURN_VOID_IF_DOUBLE typename std::enable_if_t<std::is_same<T, double>::value, void>

template<typename T, bool tail>
__global__ RETURN_VOID_IF_FLOAT MaskAndScaleGpu(uint64_t seed,
                                                one::CUDAGeneratorState* cuda_gen_state,
                                                uint64_t counter_offset, const int64_t elem_cnt,
                                                float rate, float scale, int64_t n_tail, const T* x,
                                                int8_t* mask, T* y, const T* tail_x,
                                                int8_t* tail_mask, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT =
      typename std::aligned_storage<sizeof(T) * PackFloatSize, sizeof(T) * PackFloatSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackFloatSize,
                                              sizeof(int8_t) * PackFloatSize>::type;
  RandPack4 rand_uniform_pack4;

  for (int64_t linear_index = global_thread_id * PackFloatSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackFloatSize) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);

    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    cuda::elementwise::Pack<T, PackFloatSize> x_vec;
    x_vec.storage = *x_load;

    int8_t mask_vec[PackFloatSize];
    T y_vec[PackFloatSize];
#pragma unroll
    for (int i = 0; i < PackFloatSize; i++) {
      rand_uniform_pack4.elem[i] = rand_uniform_pack4.elem[i] >= rate;
      mask_vec[i] = rand_uniform_pack4.elem[i];
      y_vec[i] = x_vec.elem[i] * mask_vec[i] * scale;
    }

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }

  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      rand_uniform_pack4.elem[i] = rand_uniform_pack4.elem[i] >= rate;
      tail_mask[i] = rand_uniform_pack4.elem[i];
      tail_y[i] = tail_x[i] * rand_uniform_pack4.elem[i] * scale;
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, bool tail>
__global__ RETURN_VOID_IF_FLOAT MaskAndScaleAddGpu(
    uint64_t seed, one::CUDAGeneratorState* cuda_gen_state, uint64_t counter_offset,
    const int64_t elem_cnt, float rate, float scale, int64_t n_tail, const T* x, int8_t* mask,
    const T* addend, T* y, const T* tail_x, int8_t* tail_mask, const T* tail_addend, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT =
      typename std::aligned_storage<sizeof(T) * PackFloatSize, sizeof(T) * PackFloatSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackFloatSize,
                                              sizeof(int8_t) * PackFloatSize>::type;

  RandPack4 rand_uniform_pack4;
  for (int64_t linear_index = global_thread_id * PackFloatSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackFloatSize) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);

    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    cuda::elementwise::Pack<T, PackFloatSize> x_vec;
    x_vec.storage = *x_load;

    const LoadT* addend_load = reinterpret_cast<const LoadT*>(&addend[linear_index]);
    cuda::elementwise::Pack<T, PackFloatSize> addend_vec;
    addend_vec.storage = *addend_load;

    int8_t mask_vec[PackFloatSize];
    T y_vec[PackFloatSize];
#pragma unroll
    for (int i = 0; i < PackFloatSize; i++) {
      mask_vec[i] = rand_uniform_pack4.elem[i];
      y_vec[i] = x_vec.elem[i] * mask_vec[i] * scale + addend_vec.elem[i];
    }

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }

  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);

#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      tail_mask[i] = rand_uniform_pack4.elem[i];
      tail_y[i] = tail_x[i] * rand_uniform_pack4.elem[i] * scale + tail_addend[i];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, bool tail>
__global__ RETURN_VOID_IF_HALF MaskAndScaleGpu(uint64_t seed,
                                               one::CUDAGeneratorState* cuda_gen_state,
                                               uint64_t counter_offset, const int64_t elem_cnt,
                                               float rate, float scale, int64_t n_tail, const T* x,
                                               int8_t* mask, T* y, const T* tail_x,
                                               int8_t* tail_mask, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT =
      typename std::aligned_storage<sizeof(half) * PackHalfSize, sizeof(half) * PackHalfSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackHalfSize,
                                              sizeof(int8_t) * PackHalfSize>::type;

  RandPack4 rand_uniform_pack4;
  half2 h2_scale = __float2half2_rn(scale);
  for (int64_t linear_index = global_thread_id * PackHalfSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackHalfSize) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    H2Pack x_vec{};
    x_vec.storage = *x_load;

    int8_t mask_vec[PackHalfSize];
    half2 y_vec[PackHalf2Size];
    half2 one_or_zero_h2[PackHalf2Size];

    mask_vec[0] = rand_uniform_pack4.elem[0] > rate;
    one_or_zero_h2[0].x = mask_vec[0];
    mask_vec[1] = rand_uniform_pack4.elem[1] > rate;
    one_or_zero_h2[0].y = mask_vec[1];
    y_vec[0] = __hmul2(__hmul2(x_vec.h2[0], one_or_zero_h2[0]), h2_scale);

    mask_vec[2] = rand_uniform_pack4.elem[2] > rate;
    one_or_zero_h2[1].x = mask_vec[2];
    mask_vec[3] = rand_uniform_pack4.elem[3] > rate;
    one_or_zero_h2[1].y = mask_vec[3];
    y_vec[1] = __hmul2(__hmul2(x_vec.h2[1], one_or_zero_h2[1]), h2_scale);

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }

  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    half half_scale = __float2half_rn(scale);
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      rand_uniform_pack4.elem[i] = rand_uniform_pack4.elem[i] > rate;
      tail_mask[i] = rand_uniform_pack4.elem[i];
      tail_y[i] = tail_x[i] * static_cast<half>(rand_uniform_pack4.elem[i]) * half_scale;
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, bool tail>
__global__ RETURN_VOID_IF_HALF MaskAndScaleAddGpu(
    uint64_t seed, one::CUDAGeneratorState* cuda_gen_state, uint64_t counter_offset,
    const int64_t elem_cnt, float rate, float scale, int64_t n_tail, const T* x, int8_t* mask,
    const T* addend, T* y, const T* tail_x, int8_t* tail_mask, const T* tail_addend, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT =
      typename std::aligned_storage<sizeof(half) * PackHalfSize, sizeof(half) * PackHalfSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackHalfSize,
                                              sizeof(int8_t) * PackHalfSize>::type;

  RandPack4 rand_uniform_pack4;
  half2 h2_scale = __float2half2_rn(scale);
  for (int64_t linear_index = global_thread_id * PackHalfSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackHalfSize) {
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    H2Pack x_vec{};
    x_vec.storage = *x_load;

    const LoadT* addend_load = reinterpret_cast<const LoadT*>(&addend[linear_index]);
    H2Pack addend_vec{};
    addend_vec.storage = *addend_load;

    int8_t mask_vec[PackHalfSize];
    half2 y_vec[PackHalf2Size];
    half2 one_or_zero_h2[PackHalf2Size];

    mask_vec[0] = rand_uniform_pack4.elem[0] > rate;
    one_or_zero_h2[0].x = mask_vec[0];
    mask_vec[1] = rand_uniform_pack4.elem[1] > rate;
    one_or_zero_h2[0].y = mask_vec[1];
    y_vec[0] =
        __hadd2(__hmul2(__hmul2(x_vec.h2[0], one_or_zero_h2[0]), h2_scale), addend_vec.h2[0]);

    mask_vec[2] = rand_uniform_pack4.elem[2] > rate;
    one_or_zero_h2[1].x = mask_vec[2];
    mask_vec[3] = rand_uniform_pack4.elem[3] > rate;
    one_or_zero_h2[1].y = mask_vec[3];
    y_vec[1] =
        __hadd2(__hmul2(__hmul2(x_vec.h2[1], one_or_zero_h2[1]), h2_scale), addend_vec.h2[0]);

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }

  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    half half_scale = __float2half_rn(scale);
    rand_uniform_pack4.storage = hiprand_uniform4(&state);
#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      rand_uniform_pack4.elem[i] = rand_uniform_pack4.elem[i] > rate;
      tail_mask[i] = rand_uniform_pack4.elem[i];
      tail_y[i] =
          tail_x[i] * static_cast<half>(rand_uniform_pack4.elem[i]) * half_scale + addend[i];
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, bool tail>
__global__ RETURN_VOID_IF_DOUBLE MaskAndScaleGpu(uint64_t seed,
                                                 one::CUDAGeneratorState* cuda_gen_state,
                                                 uint64_t counter_offset, const int64_t elem_cnt,
                                                 float rate, float scale, int64_t n_tail,
                                                 const T* x, int8_t* mask, T* y, const T* tail_x,
                                                 int8_t* tail_mask, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT = typename std::aligned_storage<sizeof(double) * PackDoubleSize,
                                              sizeof(double) * PackDoubleSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackDoubleSize,
                                              sizeof(int8_t) * PackDoubleSize>::type;
  RandPack4 rand_uniform_pack4;
  bool grid_loop_rand_state = 0;

  for (int64_t linear_index = global_thread_id * PackDoubleSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackDoubleSize) {
    if (grid_loop_rand_state == 0) {
      rand_uniform_pack4.storage = hiprand_uniform4(&state);
    } else {
      // Use the last two random numbers we generated in previous iteration.
      rand_uniform_pack4.elem[0] = rand_uniform_pack4.elem[2];
      rand_uniform_pack4.elem[1] = rand_uniform_pack4.elem[3];
      grid_loop_rand_state ^= 1;
    }
    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    cuda::elementwise::Pack<double, PackDoubleSize> x_vec;
    x_vec.storage = *x_load;

    int8_t mask_vec[PackDoubleSize];
    double y_vec[PackDoubleSize];
#pragma unroll
    for (int i = 0; i < 2; i++) {
      mask_vec[i] = rand_uniform_pack4.elem[i];
      y_vec[i] = x_vec.elem[i] * mask_vec[i] * scale;
    }

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }

  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    float single_rand_uniform = hiprand_uniform(&state);
    single_rand_uniform = single_rand_uniform > rate;
#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      tail_mask[i] = single_rand_uniform;
      tail_y[i] = tail_x[i] * single_rand_uniform * scale;
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<typename T, bool tail>
__global__ RETURN_VOID_IF_DOUBLE MaskAndScaleAddGpu(
    uint64_t seed, one::CUDAGeneratorState* cuda_gen_state, uint64_t counter_offset,
    const int64_t elem_cnt, float rate, float scale, int64_t n_tail, const T* x, int8_t* mask,
    const T* addend, T* y, const T* tail_x, int8_t* tail_mask, const T* tail_addend, T* tail_y) {
  int32_t global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, global_thread_id, cuda_gen_state->dev_offset, &state);
  using LoadT = typename std::aligned_storage<sizeof(double) * PackDoubleSize,
                                              sizeof(double) * PackDoubleSize>::type;
  using MaskT = typename std::aligned_storage<sizeof(int8_t) * PackDoubleSize,
                                              sizeof(int8_t) * PackDoubleSize>::type;

  RandPack4 rand_uniform_pack4;
  bool grid_loop_rand_state = 0;

  for (int64_t linear_index = global_thread_id * PackDoubleSize; linear_index < elem_cnt;
       linear_index += gridDim.x * blockDim.x * PackDoubleSize) {
    if (grid_loop_rand_state == 0) {
      rand_uniform_pack4.storage = hiprand_uniform4(&state);
    } else {
      // Use the last two random numbers we generated in previous iteration.
      rand_uniform_pack4.elem[0] = rand_uniform_pack4.elem[2];
      rand_uniform_pack4.elem[1] = rand_uniform_pack4.elem[3];
      grid_loop_rand_state ^= 1;
    }
    const LoadT* x_load = reinterpret_cast<const LoadT*>(x + linear_index);
    cuda::elementwise::Pack<double, PackDoubleSize> x_vec;
    x_vec.storage = *x_load;

    const LoadT* addend_load = reinterpret_cast<const LoadT*>(&addend[linear_index]);
    cuda::elementwise::Pack<double, PackDoubleSize> addend_vec;
    addend_vec.storage = *addend_load;

    int8_t mask_vec[PackDoubleSize];
    double y_vec[PackDoubleSize];
#pragma unroll
    for (int i = 0; i < PackDoubleSize; i++) {
      rand_uniform_pack4.elem[i] = rand_uniform_pack4.elem[i] > rate;
      mask_vec[i] = rand_uniform_pack4.elem[i];
      y_vec[i] = x_vec.elem[i] * mask_vec[i] * scale + addend_vec.elem[i];
    }

    *(reinterpret_cast<LoadT*>(y + linear_index)) = *reinterpret_cast<LoadT*>(y_vec);
    *(reinterpret_cast<MaskT*>(mask + linear_index)) = *reinterpret_cast<MaskT*>(mask_vec);
  }
  if (tail && global_thread_id < n_tail && global_thread_id == 0) {
    float single_rand_uniform = hiprand_uniform(&state);
    single_rand_uniform = single_rand_uniform > rate;
#pragma unroll
    for (int i = 0; i < n_tail; i++) {
      tail_mask[i] = single_rand_uniform;
      tail_y[i] = tail_x[i] * single_rand_uniform * scale + addend[i];
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int32_t new_counter = cuda::atomic::Add(&cuda_gen_state->dev_counter, 1) + 1;
    if (new_counter == gridDim.x) {
      cuda_gen_state->dev_counter = 0;               // reset counter to zero
      cuda_gen_state->dev_offset += counter_offset;  // maintain the state of generator's dev_offset
    }
  }
}

template<int pack_size>
unsigned int ComputeGridSize(const int32_t block_size, const int64_t elem_cnt) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int blocks_per_sm = prop.maxThreadsPerMultiProcessor / block_size;
  unsigned int grid_size = ((elem_cnt + block_size - 1) / block_size);
  grid_size = std::min((unsigned int)prop.multiProcessorCount * blocks_per_sm, grid_size);
  return grid_size;
}

template<typename T>
void MaskAndScale(DeviceCtx* ctx, uint64_t seed, one::CUDAGeneratorState* cuda_gen_state,
                  const int64_t elem_cnt, float rate, float scale, const T* x, int8_t* mask, T* y) {
  unsigned int grid_size = ComputeGridSize<4>(kBlockSize, elem_cnt);
  constexpr int pack_size = cuda::elementwise::PackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = pack_num - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  uint64_t counter_offset = 0;
  if (tail) {
    // If tail, we need generate randnum one more time, so here we add another `1`.
    counter_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1 + 1) * kVecSize;
    MaskAndScaleGpu<T, true><<<grid_size, kBlockSize, 0, ctx->hip_stream()>>>(
        seed, cuda_gen_state, counter_offset, elem_cnt, rate, scale, n_tail, x, mask, y,
        (x + tail_offset), (mask + tail_offset), (y + tail_offset));
  } else {
    counter_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize;
    MaskAndScaleGpu<T, false><<<grid_size, kBlockSize, 0, ctx->hip_stream()>>>(
        seed, cuda_gen_state, counter_offset, elem_cnt, rate, scale, n_tail, x, mask, y,
        (x + tail_offset), (mask + tail_offset), (y + tail_offset));
  }
}

template<typename T>
void MaskAndScaleAdd(DeviceCtx* ctx, uint64_t seed, one::CUDAGeneratorState* cuda_gen_state,
                     const int64_t elem_cnt, float rate, float scale, const T* x, int8_t* mask,
                     const T* addend, T* y) {
  unsigned int grid_size = ComputeGridSize<4>(kBlockSize, elem_cnt);
  constexpr int pack_size = cuda::elementwise::PackSize<T>();
  const int64_t pack_num = elem_cnt / pack_size;
  const int64_t tail_offset = pack_num * pack_size;
  const int64_t n_tail = pack_num - tail_offset;
  const bool tail = n_tail > 0 ? true : false;
  uint64_t counter_offset = 0;
  if (tail) {
    // If tail, we need generate randnum one more time, so here we add another `1`.
    counter_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1 + 1) * kVecSize;
    MaskAndScaleAddGpu<T, true><<<grid_size, kBlockSize, 0, ctx->hip_stream()>>>(
        seed, cuda_gen_state, counter_offset, elem_cnt, rate, scale, n_tail, x, mask, addend, y,
        (x + tail_offset), (mask + tail_offset), (addend + tail_offset), (y + tail_offset));
  } else {
    counter_offset = ((elem_cnt - 1) / (kBlockSize * grid_size * kVecSize) + 1) * kVecSize;
    MaskAndScaleAddGpu<T, false><<<grid_size, kBlockSize, 0, ctx->hip_stream()>>>(
        seed, cuda_gen_state, counter_offset, elem_cnt, rate, scale, n_tail, x, mask, addend, y,
        (x + tail_offset), (mask + tail_offset), (addend + tail_offset), (y + tail_offset));
  }
}

template<typename T>
struct MaskAndScaleFunctor {
  OF_DEVICE_FUNC explicit MaskAndScaleFunctor(float scale) : scale(scale) {}
  OF_DEVICE_FUNC T operator()(T x, int8_t mask) const {
    return x * static_cast<T>(mask) * static_cast<T>(scale);
  }
  float scale;
};

template<typename T>
class DropoutKernelGPU final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  DropoutKernelGPU() = default;
  ~DropoutKernelGPU() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(DeviceType::kGPU));
    return std::make_shared<FusedDropoutKernelState>(generator);
  }

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    auto* fused_dropout_kernel_state = dynamic_cast<FusedDropoutKernelState*>(state);
    CHECK_NOTNULL(fused_dropout_kernel_state);
    const auto& generator = fused_dropout_kernel_state->generator();
    CHECK_NOTNULL(generator);
    std::shared_ptr<one::CUDAGeneratorImpl> cuda_generator =
        CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>());
    uint64_t seed = cuda_generator->current_seed();

    const float rate = ctx->Attr<float>("rate");
    float scale = 0.0;
    if (rate < 1.0f) { scale = 1.0f / (1.0f - rate); }
    one::CUDAGeneratorState* cuda_gen_state = cuda_generator->cuda_gen_state();

    if (ctx->has_input("_add_to_output", 0)) {
      const user_op::Tensor* addend = ctx->Tensor4ArgNameAndIndex("_add_to_output", 0);
      MaskAndScaleAdd<T>(ctx->device_ctx(), seed, cuda_gen_state, in->shape().elem_cnt(), rate,
                         scale, in->dptr<T>(), mask->mut_dptr<int8_t>(), addend->dptr<T>(),
                         out->mut_dptr<T>());
    } else {
      MaskAndScale<T>(ctx->device_ctx(), seed, cuda_gen_state, in->shape().elem_cnt(), rate, scale,
                      in->dptr<T>(), mask->mut_dptr<int8_t>(), out->mut_dptr<T>());
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_KERNEL_GPU(dtype)                                                \
  REGISTER_USER_KERNEL("dropout").SetCreateFn<DropoutKernelGPU<dtype>>().SetIsMatchedHob( \
      (user_op::HobDeviceType() == DeviceType::kGPU)                                      \
      && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value)                    \
      && (user_op::HobDataType("mask", 0) == GetDataType<int8_t>::value));

REGISTER_DROPOUT_KERNEL_GPU(half)
REGISTER_DROPOUT_KERNEL_GPU(float)
REGISTER_DROPOUT_KERNEL_GPU(double)

template<typename T>
class DropoutGradKernelGPU final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  DropoutGradKernelGPU() = default;
  ~DropoutGradKernelGPU() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const float scale = ctx->Attr<float>("scale");
    const int64_t elem_cnt = dy->shape().elem_cnt();
    OF_CUDA_CHECK((cuda::elementwise::Binary(MaskAndScaleFunctor<T>(scale), elem_cnt,
                                             dx->mut_dptr<T>(), dy->dptr<T>(), mask->dptr<int8_t>(),
                                             ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DROPOUT_GRAD_KERNEL_GPU(dtype)                                                 \
  REGISTER_USER_KERNEL("dropout_grad")                                                          \
      .SetCreateFn<DropoutGradKernelGPU<dtype>>()                                               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kGPU)                           \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value))         \
      .SetInplaceProposalFn([](const user_op::InferContext&,                                    \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("dx", 0, "dy", 0, true));                        \
        return Maybe<void>::Ok();                                                               \
      });

REGISTER_DROPOUT_GRAD_KERNEL_GPU(half)
REGISTER_DROPOUT_GRAD_KERNEL_GPU(float)
REGISTER_DROPOUT_GRAD_KERNEL_GPU(double)

}  // namespace

}  // namespace oneflow
