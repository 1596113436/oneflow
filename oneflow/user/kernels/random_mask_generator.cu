#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/random_mask_generator.h"

namespace oneflow {

namespace {

constexpr int32_t kMinPackPerThread = 2;

// using PackType = ulonglong2;
using PackType = int32_t;

union Pack {
  PackType p_value;
  int8_t b_value[sizeof(PackType)];
};

// __device__ int8_t GenMask(hiprandState* state, const float rate) {
//   return hiprand_uniform(state) >= rate;
// }

// __global__ void GenerateGpu(hiprandState* state, const int64_t n, const float rate, int8_t* mask) {
//   const int id = blockIdx.x * blockDim.x + threadIdx.x;
//   hiprandState localState = state[id];
//   PackType* pack_mask = reinterpret_cast<PackType*>(mask);
//   Pack pack;
//   CUDA_1D_KERNEL_LOOP(i, n / sizeof(PackType)) {
// #pragma unroll
//     for (int j = 0; j < sizeof(PackType); ++j) { pack.b_value[j] = GenMask(&localState, rate); }
//     pack_mask[i] = pack.p_value;
//   }
//   const int32_t rem_cnt = n % sizeof(PackType);
//   const int32_t rem_offset = n - rem_cnt;
//   if (id < rem_cnt) { mask[id + rem_offset] = GenMask(&localState, rate); }
//   state[id] = localState;
// }

// }  // namespace

// void RandomMaskGenerator<DeviceType::kGPU>::Generate(DeviceCtx* device_ctx, const int64_t n,
//                                                      const float rate, int8_t* mask) {
//   int32_t block_num = generator_->max_block_num();
//   int32_t thread_num = generator_->max_thread_num();
//   auto* curand_states = generator_->curand_states();
//   const int32_t elem_cnt_per_block = thread_num * sizeof(PackType) * kMinPackPerThread;
//   const int32_t block_num_final =
//       std::min(static_cast<int32_t>((n + elem_cnt_per_block - 1) / elem_cnt_per_block), block_num);
//   GenerateGpu<<<block_num_final, thread_num, 0, device_ctx->hip_stream()>>>(curand_states, n, rate,
//                                                                              mask);
// }

// template class RandomMaskGenerator<DeviceType::kGPU>;


__global__ void GenerateGpu(one::PhiloxCUDAState philox_args, const int64_t n, const float rate, int8_t* mask) {
//   const int id = blockIdx.x * blockDim.x + threadIdx.x;
//   hiprandState localState = state[id];
//   PackType* pack_mask = reinterpret_cast<PackType*>(mask);
//   Pack pack;
//   CUDA_1D_KERNEL_LOOP(i, n / sizeof(PackType)) {
// #pragma unroll
//     for (int j = 0; j < sizeof(PackType); ++j) { pack.b_value[j] = GenMask(&localState, rate); }
//     pack_mask[i] = pack.p_value;
//   }
//   const int32_t rem_cnt = n % sizeof(PackType);
//   const int32_t rem_offset = n - rem_cnt;
//   if (id < rem_cnt) { mask[id + rem_offset] = GenMask(&localState, rate); }
//   state[id] = localState;
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    hiprandStatePhilox4_32_10_t state; 
    // auto seeds = at::cuda::philox::unpack(philox_args);
    hiprand_init(0, index, 0, &state); 
    PackType* pack_mask = reinterpret_cast<PackType*>(mask);
    Pack pack;
    CUDA_1D_KERNEL_LOOP(i, n / sizeof(PackType)) {
        float4 rand = hiprand_uniform4(&state);
    // #pragma unroll
        // for (int j = 0; j < sizeof(PackType); ++j) { 
            pack.b_value[0] = rand.x > rate;
            pack.b_value[1] = rand.y > rate; 
            pack.b_value[2] = rand.z > rate; 
            pack.b_value[3] = rand.w > rate; 
        // }
        pack_mask[i] = pack.p_value;
    }
}

}  // namespace

void RandomMaskGenerator<DeviceType::kGPU>::Generate(DeviceCtx* device_ctx, const int64_t n,
                                                     const float rate, int8_t* mask) {
  int32_t block_num = generator_->max_block_num();
  int32_t thread_num = generator_->max_thread_num();
  auto* curand_states = generator_->curand_states();
  const int32_t elem_cnt_per_block = thread_num * sizeof(PackType) * kMinPackPerThread;
  const int32_t block_num_final =
      std::min(static_cast<int32_t>((n + elem_cnt_per_block - 1) / elem_cnt_per_block), block_num);
//   GenerateGpu<<<block_num_final, thread_num, 0, device_ctx->hip_stream()>>>(curand_states, n, rate,
//                                                                              mask);
  int32_t UNROLL = 4; 
  int32_t block_size = 256; 
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop,0);
  unsigned int grid_size = ((n + block_size -1) / block_size);
  unsigned int blocks_per_sm = prop.maxThreadsPerMultiProcessor/block_size;
  grid_size = std::min((unsigned int)prop.multiProcessorCount * blocks_per_sm, grid_size);
  int64_t counter_offset = ((n - 1)/(block_size*grid_size*UNROLL)+1)*UNROLL;
//   std::lock_guard<std::mutex> lock(generator_->mutex_);
  one::PhiloxCUDAState rng_engine_inputs = generator_->philox_cuda_state(counter_offset);
  printf("Grid size is: %u \n", grid_size); 
  printf("Block size is: %u \n", block_size); 
  GenerateGpu<<<grid_size, block_size, 0, device_ctx->hip_stream()>>>(rng_engine_inputs, n, rate, mask);
}

template class RandomMaskGenerator<DeviceType::kGPU>;



}  // namespace oneflow
