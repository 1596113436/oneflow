#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/primitive/include/broadcast_elementwise_binary.h"
#include "oneflow/core/primitive/common/broadcast_elementwise_binary.h"
#include "oneflow/core/primitive/cuda/type_seq.h"
#include "oneflow/core/stream/cuda/cuda_stream_context.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/core/primitive/cuda/binary_functor.cuh"

namespace oneflow {

namespace primitive {

namespace {

template<BinaryOp binary_op, typename Src, typename Dst, size_t num_dims, size_t src0_pack_size,
         size_t src1_pack_size, typename IndexType>
__global__ void BroadcastElementwiseBinaryGpu(
    BroadcastElementwiseBinaryParams<num_dims, IndexType> params) {
  constexpr size_t dst_pack_size =
      src0_pack_size > src1_pack_size ? src0_pack_size : src1_pack_size;
  static_assert(src0_pack_size == dst_pack_size || src0_pack_size == 1, "");
  static_assert(src1_pack_size == dst_pack_size || src1_pack_size == 1, "");

  const PackType<Src, src0_pack_size>* src0 =
      reinterpret_cast<const PackType<Src, src0_pack_size>*>(params.src0);
  const PackType<Src, src1_pack_size>* src1 =
      reinterpret_cast<const PackType<Src, src1_pack_size>*>(params.src1);
  PackType<Dst, dst_pack_size>* dst = reinterpret_cast<PackType<Dst, dst_pack_size>*>(params.dst);

  IndexType src0_index[num_dims];
  IndexType src1_index[num_dims];
  IndexType dst_index[num_dims];
  CUDA_1D_KERNEL_LOOP_T(IndexType, offset, params.count) {
    params.dst_index_helper.OffsetToNdIndex(offset, dst_index);
    for (int64_t i = 0; i < num_dims; ++i) {
      if (params.src0_dims[i] == 1) {
        src0_index[i] = 0;
      } else {
        src0_index[i] = dst_index[i];
      }
      if (params.src1_dims[i] == 1) {
        src1_index[i] = 0;
      } else {
        src1_index[i] = dst_index[i];
      }
    }
    const IndexType src0_offset = params.src0_index_helper.NdIndexToOffset(src0_index);
    const IndexType src1_offset = params.src1_index_helper.NdIndexToOffset(src1_index);
    Pack<Src, src0_pack_size> src0_pack;
    src0_pack.storage = src0[src0_offset];
    Pack<Src, src1_pack_size> src1_pack;
    src1_pack.storage = src1[src1_offset];
    Pack<Dst, dst_pack_size> dst_pack;

#pragma unroll
    for (int j = 0; j < dst_pack_size; ++j) {
      const Src src0_val =
          (src0_pack_size == dst_pack_size) ? src0_pack.elem[j] : src0_pack.elem[0];
      const Src src1_val =
          (src1_pack_size == dst_pack_size) ? src1_pack.elem[j] : src1_pack.elem[0];
      dst_pack.elem[j] = BinaryFunctor<DeviceType::kGPU, binary_op, Src, Dst>()(src0_val, src1_val);
    }
    dst[offset] = dst_pack.storage;
  }
}

template<BinaryOp op, typename Src, typename Dst, size_t num_dims, size_t src0_pack_size,
         size_t src1_pack_size, typename IndexType>
void LaunchKernel(StreamContext* stream_ctx,
                  BroadcastElementwiseBinaryParams<num_dims, IndexType> params) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
  BroadcastElementwiseBinaryGpu<op, Src, Dst, num_dims, src0_pack_size, src1_pack_size, IndexType>
      <<<BlocksNum4ThreadsNum(params.count), kCudaThreadsNumPerBlock, 0, hip_stream>>>(params);
}

template<BinaryOp binary_op, typename Src, typename Dst>
struct BinaryLhsScalarFunctor {
  __host__ __device__ explicit BinaryLhsScalarFunctor(Src scalar) : scalar(scalar) {}
  __device__ Dst operator()(Src src) const {
    return BinaryFunctor<DeviceType::kGPU, binary_op, Src, Dst>()(scalar, src);
  }
  const Src scalar;
};

template<BinaryOp binary_op, typename Src, typename Dst>
struct BinaryRhsScalarFunctor {
  __host__ __device__ explicit BinaryRhsScalarFunctor(Src scalar) : scalar(scalar) {}
  __device__ Dst operator()(Src src) const {
    return BinaryFunctor<DeviceType::kGPU, binary_op, Src, Dst>()(src, scalar);
  }
  const Src scalar;
};

template<BinaryOp binary_op, typename Src, typename Dst>
struct BinaryLhsScalarPtrFunctorFactory {
  __host__ __device__ explicit BinaryLhsScalarPtrFunctorFactory(const Src* scalar_ptr)
      : scalar_ptr(scalar_ptr) {}
  __device__ BinaryLhsScalarFunctor<binary_op, Src, Dst> operator()() const {
    return BinaryLhsScalarFunctor<binary_op, Src, Dst>(*scalar_ptr);
  }
  const Src* scalar_ptr;
};

template<BinaryOp binary_op, typename Src, typename Dst>
struct BinaryRhsScalarPtrFunctorFactory {
  __host__ __device__ explicit BinaryRhsScalarPtrFunctorFactory(const Src* scalar_ptr)
      : scalar_ptr(scalar_ptr) {}
  __device__ BinaryRhsScalarFunctor<binary_op, Src, Dst> operator()() const {
    return BinaryRhsScalarFunctor<binary_op, Src, Dst>(*scalar_ptr);
  }
  const Src* scalar_ptr;
};

bool IsDimsEquals(size_t num_src0_dims, const int64_t* src0_dims, size_t num_src1_dims,
                  const int64_t* src1_dims) {
  if (num_src0_dims != num_src1_dims) { return false; }
  for (size_t i = 0; i < num_src1_dims; ++i) {
    if (src0_dims[i] != src1_dims[i]) { return false; }
  }
  return true;
}

template<BinaryOp binary_op, typename Src, typename Dst>
void DispatchLaunch(StreamContext* stream_ctx, size_t num_src0_dims, const int64_t* src0_dims,
                    const Src* src0, size_t num_src1_dims, const int64_t* src1_dims,
                    const Src* src1, Dst* dst) {
  hipStream_t hip_stream =
      CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
  size_t src0_count = GetElementCount(num_src0_dims, src0_dims);
  size_t src1_count = GetElementCount(num_src1_dims, src1_dims);
  const size_t elem_cnt = std::max(src0_count, src1_count);
  if (IsDimsEquals(num_src0_dims, src0_dims, num_src1_dims, src1_dims)) {
    OF_CUDA_CHECK((cuda::elementwise::Binary(BinaryFunctor<DeviceType::kGPU, binary_op, Src, Dst>(),
                                             elem_cnt, dst, src0, src1, hip_stream)));
  } else if (src0_count == 1) {
    OF_CUDA_CHECK((cuda::elementwise::UnaryWithFactory(
        BinaryLhsScalarPtrFunctorFactory<binary_op, Src, Dst>(src0), elem_cnt, dst, src1,
        hip_stream)));
  } else if (src1_count == 1) {
    OF_CUDA_CHECK((cuda::elementwise::UnaryWithFactory(
        BinaryRhsScalarPtrFunctorFactory<binary_op, Src, Dst>(src1), elem_cnt, dst, src0,
        hip_stream)));
  } else {
    SimplifyThenLaunch<binary_op, Src, Dst>(stream_ctx, num_src0_dims, src0_dims, src0,
                                            num_src1_dims, src1_dims, src1, dst);
  }
}

template<typename T>
T GetValue(Scalar value) {
  return value.Value<T>();
}

template<>
half GetValue<half>(Scalar value) {
  return static_cast<half>(GetValue<float>(value));
}

#if CUDA_VERSION >= 11000

template<>
hip_bfloat16 GetValue<hip_bfloat16>(Scalar value) {
  return static_cast<hip_bfloat16>(GetValue<float>(value));
}

#endif  // CUDA_VERSION >= 11000

template<BinaryOp binary_op, typename Src, typename Dst>
class BroadcastElementwiseBinaryImpl : public BroadcastElementwiseBinary {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BroadcastElementwiseBinaryImpl);
  BroadcastElementwiseBinaryImpl() = default;
  ~BroadcastElementwiseBinaryImpl() override = default;

  void Launch(StreamContext* stream_ctx, Scalar src0, size_t num_src1_dims,
              const int64_t* src1_dims, const void* src1, void* dst) override {
    hipStream_t hip_stream =
        CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
    const size_t elem_cnt = GetElementCount(num_src1_dims, src1_dims);
    OF_CUDA_CHECK((cuda::elementwise::Unary(
        BinaryLhsScalarFunctor<binary_op, Src, Dst>(GetValue<Src>(src0)), elem_cnt,
        reinterpret_cast<Dst*>(dst), reinterpret_cast<const Src*>(src1), hip_stream)));
  }
  void Launch(StreamContext* stream_ctx, size_t num_src0_dims, const int64_t* src0_dims,
              const void* src0, Scalar src1, void* dst) override {
    hipStream_t hip_stream =
        CHECK_NOTNULL(dynamic_cast<CudaStreamContext*>(stream_ctx))->hip_stream();
    const size_t elem_cnt = GetElementCount(num_src0_dims, src0_dims);
    OF_CUDA_CHECK((cuda::elementwise::Unary(
        BinaryRhsScalarFunctor<binary_op, Src, Dst>(GetValue<Src>(src1)), elem_cnt,
        reinterpret_cast<Dst*>(dst), reinterpret_cast<const Src*>(src0), hip_stream)));
  }
  void Launch(StreamContext* stream_ctx, size_t num_src0_dims, const int64_t* src0_dims,
              const void* src0, size_t num_src1_dims, const int64_t* src1_dims, const void* src1,
              void* dst) override {
    DispatchLaunch<binary_op, Src, Dst>(
        stream_ctx, num_src0_dims, src0_dims, reinterpret_cast<const Src*>(src0), num_src1_dims,
        src1_dims, reinterpret_cast<const Src*>(src1), reinterpret_cast<Dst*>(dst));
  }
};

template<BinaryOp binary_op, typename Src, typename Dst>
std::unique_ptr<BroadcastElementwiseBinary> NewBroadcastElementwiseBinary() {
  return std::unique_ptr<BroadcastElementwiseBinary>(
      new BroadcastElementwiseBinaryImpl<binary_op, Src, Dst>());
}

class BroadcastElementwiseBinaryFactoryImpl : public BroadcastElementwiseBinaryFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(BroadcastElementwiseBinaryFactoryImpl);
  BroadcastElementwiseBinaryFactoryImpl() = default;
  ~BroadcastElementwiseBinaryFactoryImpl() override = default;

  std::unique_ptr<BroadcastElementwiseBinary> New(BinaryOp binary_op, DataType src_type,
                                                  DataType dst_type, size_t max_num_dims) override {
    if (max_num_dims > kMaxNumDims) { return nullptr; }
#define MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY(binary_op, data_type_pair) \
  {std::make_tuple(binary_op, OF_PP_PAIR_SECOND(data_type_pair),                    \
                   OF_PP_PAIR_SECOND(data_type_pair)),                              \
   NewBroadcastElementwiseBinary<binary_op, OF_PP_PAIR_FIRST(data_type_pair),       \
                                 OF_PP_PAIR_FIRST(data_type_pair)>},

#define MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY(      \
    binary_op, src_data_type_pair, dst_data_type_pair)                            \
  {std::make_tuple(binary_op, OF_PP_PAIR_SECOND(src_data_type_pair),              \
                   OF_PP_PAIR_SECOND(dst_data_type_pair)),                        \
   NewBroadcastElementwiseBinary<binary_op, OF_PP_PAIR_FIRST(src_data_type_pair), \
                                 OF_PP_PAIR_FIRST(dst_data_type_pair)>},

    static const std::map<std::tuple<BinaryOp, DataType, DataType>,
                          std::function<std::unique_ptr<BroadcastElementwiseBinary>()>>
        new_broadcast_elementwise_binary_handle{
            OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY,
                                             BINARY_MATH_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ)
                OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
                    MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY,
                    BINARY_COMPARISION_OP_SEQ BINARY_LOGICAL_OP_SEQ, CUDA_PRIMITIVE_ALL_TYPE_SEQ,
                    CUDA_PRIMITIVE_INT8_TYPE_SEQ)};

#undef MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_COMPARASION_AND_LOGICAL_ENTRY
#undef MAKE_NEW_BROADCAST_ELEMENTWISE_BINARY_MATH_ENTRY

    const auto it = new_broadcast_elementwise_binary_handle.find(
        std::make_tuple(binary_op, src_type, dst_type));
    if (it != new_broadcast_elementwise_binary_handle.end()) {
      return it->second();
    } else {
      return nullptr;
    }
  }
};

REGISTER_PRIMITIVE_FACTORY(DeviceType::kGPU, BroadcastElementwiseBinaryFactory,
                           BroadcastElementwiseBinaryFactoryImpl);

}  // namespace

}  // namespace primitive

}  // namespace oneflow
